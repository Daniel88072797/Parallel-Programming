/*shared memory、Coalesced Memory Access、sequential addressing(handle bank conflict、handle warp divergent)、curand()、double->float*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
// #include "parameters.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#define DEV_NO 0
#define c1 2
#define c2 2
#define w 1
#define sizepop 4096
#define popmax 100
#define popmin -100
#define Vmax (popmax-popmin)
#define Vmin -(popmax-popmin)
#define gen 1000000
#define r (float)rand() / RAND_MAX  //-1 + 2 * ((float)rand()) / RAND_MAX
#define poprange popmax - popmin
// #define func(a) (((a)*(a)*(a)))
#define func(x) fabs(8000.0 + x * (-10000.0 + x * (-0.8 + x)));
hipDeviceProp_t prop;
float *pop=new float[sizepop];
float *V=new float[sizepop];
float *fitness=new float[sizepop];
float *pbest=new float[sizepop];
float *gbest=new float[1];
float *outputMax=new float[sizepop];
// float *answer=new float[1];
int best_index=0;
float *fitnesspbest=new float[sizepop];
float *fitnessgbest=new float[1];
// float elapsedTime;

// float func(float x) { return fabs(8000.0 + x * (-10000.0 + x * (-0.8 + x))); }

/*初始化粒子的位置和速度*/
void pop_init(float *pop){

        // #pragma unroll 32
        for(int i = 0; i < sizepop; i++){  

                pop[i] = r * poprange + popmin;  //position
                V[i] =  r * Vmax; 
                fitness[i] = func(pop[i]);   //value

        }
}

/*找出最大目標值*/
void max(float *fit,int size){
        int index = 0;
        float max = *fit; //set max value is first
     
        for(int i = 1;i < size; i++){
                if(*(fit+i) > max){
                    max = *(fit + i);
                    index = i;
                }
        }

        printf("max : %f\n",max);
        best_index = index;
        *fitnessgbest = max;
        outputMax[0] = max;
        
}


/*首先初始化粒子的位置和速度並求得「粒子自身最佳位置」、「群體最佳位置」、「個體最佳解」、「群體最佳解」*/
void PSO_init(void)
{
        pop_init(pop);
        max(fitness,sizepop);
        
        *gbest = pop[best_index];
        
        for(int i = 0; i < sizepop; i++){
                
                pbest[i] = pop[i];
                pbest[i] = pop[i];
                
        }
        for(int i = 0; i < sizepop; i++){
                fitnesspbest[i] = fitness[i];
        }

}

__global__ void init_kernel(hiprandState *state, long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    hiprand_init(seed, idx, 0, state);
}


__global__ void kernel1(float *V, float *pop, float *pbest, float *gbest, hiprandState *state, float *fitness, float *fitnesspbest, float *outputMax){   //改用cuRand
        
        extern __shared__ float sdata[];
        int pno = blockIdx.x * blockDim.x +  threadIdx.x;
        int tid = threadIdx.x;
        
        
        if(fitnesspbest[pno] == outputMax[0]) *gbest = pbest[pno]; 
        __syncthreads(); 

        float r1 = hiprand_uniform(state);
        float r2 = hiprand_uniform(state);
        // printf("random: %lf  %lf\n", r1, r2);
        // if(tid == 0) printf("thread %d gbest: %lf\n",tid, *gbest);

        // printf(" %lf   %lf\n",random1,random2);
        // printf("V: %lf\n",V[i]);
        // printf("pno: %d \n",pno);
        /*update valocity*/
        V[pno] = w * V[pno] + c1 * (r1) * (pbest[pno] - pop[pno]) + c2 * (r2) * (*gbest - pop[pno]);   

        if (V[pno] > Vmax) V[pno] = Vmax;
        if (V[pno] < Vmin) V[pno] = Vmin;        

        // printf("pop: %lf\n",pop[i]);

        /*update position*/
        pop[pno] = pop[pno] + V[pno];
        
        if(pop[pno] > popmax) pop[pno] = popmax;
        if(pop[pno] < popmin) pop[pno] = popmin;

        /*update fitness*/
        fitness[pno] = func(pop[pno]); 

        /*update pbest、fitnesspbest*/
        if(fitness[pno] > fitnesspbest[pno]){       
                pbest[pno] = pop[pno];   
                fitnesspbest[pno] = fitness[pno]; 
        } 

        /*move fitnesspbest to shared memory data*/
        sdata[tid] = fitnesspbest[pno];
        __syncthreads();

        #pragma unroll 5
        for(int s = blockDim.x / 2; s > 0; s >>= 1){
                if(tid < s){
                        sdata[tid] = max(sdata[tid], sdata[tid + s]); 
                }
                __syncthreads();
        }



        if(tid == 0) outputMax[blockIdx.x] = sdata[tid]; 



        
        
}
__global__ void kernel2(float *outputMax){   //利用reduction

        extern __shared__ float srdata[];
        int tid = threadIdx.x;
        srdata[tid] = outputMax[tid];
        __syncthreads();

        /*second reduction*/
        for(int s = blockDim.x / 2; s > 0; s >>= 1){
                if(tid < s){
                        srdata[tid] = max(srdata[tid], srdata[tid + s]); 
                }
                __syncthreads();
        }

        if(tid == 0) outputMax[tid] = srdata[tid];
        // if(tid == 0) printf("fitness gbest per iter: %lf\n", outputMax[0]); 

}

int main()
{       
        /*count time*/
        struct timespec start, end, temp;
        float time_used;
        clock_gettime(CLOCK_MONOTONIC, &start);

        hipGetDeviceProperties(&prop, DEV_NO);
        printf("maxThreadsPerBlock = %d\nsharedMemPerBlock = %lu\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);   //Shared memory available per block in bytes
        
        PSO_init(); 

	float *d_pop,*d_V,*d_fitnesspbest,*d_pbest;
        float *d_gbest,*d_fitnessgbest,*d_fitness;
        float *d_outputMax;
        
        hiprandState *state; 
        hipMalloc(&state, sizeof(hiprandState)); 
        init_kernel<<<1,1>>>(state, clock());         
        


	// Allocate Memory Space on Device
        hipMalloc(&d_fitness,sizeof(float)*sizepop);
        hipMalloc(&d_pop,sizeof(float)*sizepop);
        hipMalloc(&d_V,sizeof(float)*sizepop);
        hipMalloc(&d_fitnesspbest,sizeof(float)*sizepop);
        hipMalloc(&d_pbest,sizeof(float)*sizepop);
        hipMalloc(&d_gbest,sizeof(float));
        hipMalloc(&d_fitnessgbest,sizeof(float));
        
        hipMalloc(&d_outputMax,sizeof(float) * ceil(((float)sizepop) / 1024));
 
	// Copy Data to be Calculated
        hipMemcpy(d_fitness, fitness, sizeof(float)*sizepop, hipMemcpyHostToDevice);
        hipMemcpy(d_pop, pop, sizeof(float)*sizepop, hipMemcpyHostToDevice);
        hipMemcpy(d_V, V, sizeof(float)*sizepop, hipMemcpyHostToDevice);
        hipMemcpy(d_fitnesspbest, fitnesspbest, sizeof(float)*sizepop, hipMemcpyHostToDevice);
        hipMemcpy(d_pbest, pbest, sizeof(float)*sizepop, hipMemcpyHostToDevice);
        hipMemcpy(d_gbest, gbest, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_fitnessgbest, fitnessgbest, sizeof(float), hipMemcpyHostToDevice);

        hipMemcpy(d_outputMax, outputMax, sizeof(float), hipMemcpyHostToDevice); 


        int num_block = ceil(((float)sizepop) / 1024);
        printf("number of block: %d\n",num_block);

        int num_threads = (sizepop < 1024)? sizepop : 1024;
        printf("number of threads: %d\n",num_threads);

        for(int i = 0; i < gen; i++){

                kernel1<<<num_block ,num_threads, num_threads * sizeof(float)>>>(d_V, d_pop, d_pbest, d_gbest, state, d_fitness, d_fitnesspbest, d_outputMax);
                kernel2<<<1, num_block, num_block * sizeof(float)>>>(d_outputMax);

        }

        /*final answer*/
        float answer;
        hipMemcpy(&answer, d_outputMax, sizeof(float), hipMemcpyDeviceToHost); 
        printf("The answer is: %f\n", answer);

        /*等待kernel執行完*/
        // cudaDeviceSynchronize();

        /*count time*/
        clock_gettime(CLOCK_MONOTONIC, &end);
        if ((end.tv_nsec - start.tv_nsec) < 0) {
        temp.tv_sec = end.tv_sec-start.tv_sec-1;
        temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
        } else {
        temp.tv_sec = end.tv_sec - start.tv_sec;
        temp.tv_nsec = end.tv_nsec - start.tv_nsec;
        }
        time_used = temp.tv_sec + (float) temp.tv_nsec / 1000000000.0;
        
        printf("%f second\n", time_used);

}
