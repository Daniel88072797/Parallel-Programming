#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
// #include "parameters.h"
#include <hip/hip_runtime.h>
#define DEV_NO 0
#define c1 2
#define c2 2
#define w 1
#define sizepop 2048
#define popmax 100
#define popmin -100
#define Vmax (popmax-popmin)
#define Vmin -(popmax-popmin)
#define gen 500000
#define r (double)rand() / RAND_MAX  //-1 + 2 * ((double)rand()) / RAND_MAX
#define poprange popmax - popmin
// #define func(a) (((a)*(a)*(a)))

hipDeviceProp_t prop;
double *pop=new double[sizepop];
double *V=new double[sizepop];
double *fitness=new double[sizepop];
double result[gen];
double *pbest=new double[sizepop];
double *gbest=new double[1];
int best_index=0;
double *fitnesspbest=new double[sizepop];
double *fitnessgbest=new double[1];
double genbest[gen];
// double elapsedTime;

double func(double x) { return fabs(8000.0 + x * (-10000.0 + x * (-0.8 + x))); }

// extern double GPU_kernel(double *pop1, double *V1,double *fitnesspbest1,double *pbest1,double *gbest1,double *fitnessgbest1,double r1);

/*copy*/
void function_1(double B[],double A[]){
	for(int i=0;i<sizepop;i++){
		B[i]=A[i];
	}
}


/*初始化粒子的位置和速度*/
void pop_init(double *pop){
        for(int i = 0; i < sizepop; i++){  
                pop[i] = r * poprange + popmin;  //position
                V[i] =  r * Vmax; 
                fitness[i] = func(pop[i]);   //value
                // printf("pop: %lf\n",pop[i]);
                // printf("V: %lf\n",V[i]);
                // printf("fitness: %lf\n",fitness[i]);
        }
        
}

/*找出最大目標值*/
void max(double *fit,int size){
        int index = 0;
        double max = *fit; //set max value is first
     
        for(int i = 1;i < size; i++){
                if(*(fit+i) > max){
                    max = *(fit + i);
                    index = i;
                }

        }
        printf("max : %f\n",max);
        best_index = index;
        *fitnessgbest = max;
        
}
int final_bestfitness(double *fit,int size){
        int index = 0;
        double max = *fit; 
        for(int i = 1;i < size; i++){
                if(*(fit + i) > max)
                {
                        max = *(fit + i);
                        index = i;
                }

                // printf("%lf\n", *(fit+i));
        }

        return index;
        
}

/*更新粒子的位置和速度*/
void updateVelocity(double *V ){
        for(int i = 0;i < sizepop; i++){
                // printf("V: %lf\n",V[i]);
	        V[i] = w*V[i]+c1*(r)*(pbest[i]-pop[i])+(c2)*(r)*(*gbest-pop[i]);   

                if (V[i] > Vmax) V[i] = Vmax;
                if (V[i] < Vmin) V[i] = Vmin;                     
        }

}
void updatePosition(double *pop){           
         for(int i = 0;i < sizepop; i++){
                //  printf("pop: %lf\n",pop[i]);
                 pop[i] = pop[i] + V[i];
                 
                 if(pop[i] > popmax) pop[i] = popmax;
                 if(pop[i] < popmin) pop[i] = popmin; 
          }                     

}


/*首先初始化粒子的位置和速度並求得「粒子自身最佳位置」、「群體最佳位置」、「個體最佳解」、「群體最佳解」*/
void PSO_init(void)
{
        pop_init(pop);
        max(fitness,sizepop);
        
        *gbest = pop[best_index];
        
        for(int i = 0; i < sizepop; i++){
                
                pbest[i] = pop[i];
                
        }
        for(int i = 0; i < sizepop; i++){
                fitnesspbest[i] = fitness[i];
        }

}

/*更新粒子的位置和速度、找出每一iteration的最佳位置及最佳解*/
void PSO(void){ 
        for(int i = 0;i < gen; i++){
                updateVelocity( V );
                updatePosition( pop );
                for(int j = 0; j < sizepop; j++){
                        // printf("pop %d :%lf ",j,pop[j]);
                        // printf("fitness %d :%lf ",j,fitness[j]);
                        fitness[j] = func(pop[j]); //update fitness
                        // printf("%lf ",fitness[j]);
                }                            
                // printf("\n");
                for(int j = 0; j < sizepop; j++){
                        if(fitness[j] > fitnesspbest[j]){
                                pbest[j] = pop[j];
                                fitnesspbest[j] = fitness[j]; 
                        } 
                        if(fitness[j] > *fitnessgbest){
                                *gbest = pop[j];
                                *fitnessgbest = fitness[j];
                        }   
                }
               
                genbest[i]= *gbest;
                result[i] = *fitnessgbest;

        }
        
}
int main()
{       
        /*count time*/
        struct timespec start, end, temp;
        double time_used;
        clock_gettime(CLOCK_MONOTONIC, &start);

        hipGetDeviceProperties(&prop, DEV_NO);
        printf("maxThreadsPerBlock = %d\nsharedMemPerBlock = %lu\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);   //Shared memory available per block in bytes

        // srand(5);
        PSO_init(); 
        //gpu
        PSO(); 

        int best_gen_number = final_bestfitness(result,gen);
        // for(int i = 0; i < gen; i++){
        //     printf("result %lf\n",result[i]);
        // }
        // printf("best_gen_number %d",best_gen_number);
        // for(int i = 0; i < gen; i++){
        //         printf("genbest[%d] = %lf\n", i, genbest[i]);

        // }
        /*找出最後的最佳解*/
        printf("bestposition is (%lf).\n",genbest[best_gen_number]);
        printf("bestfitness for CPU : %lf.\n",result[best_gen_number]);

        // end = clock();
      
        // printf("CPU time : %5.2f ms\n",(double)(end,start)/1000);

        /*Please press any key to exit the program*/
        // getchar();


        /*count time*/
        clock_gettime(CLOCK_MONOTONIC, &end);
        if ((end.tv_nsec - start.tv_nsec) < 0) {
        temp.tv_sec = end.tv_sec-start.tv_sec-1;
        temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
        } else {
        temp.tv_sec = end.tv_sec - start.tv_sec;
        temp.tv_nsec = end.tv_nsec - start.tv_nsec;
        }
        time_used = temp.tv_sec + (double) temp.tv_nsec / 1000000000.0;
        
        printf("%f second\n", time_used);

}
