/*shared memory、Coalesced Memory Access、interleaved  addressing(handle warp divergent)*/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
// #include "parameters.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#define DEV_NO 0
#define c1 2
#define c2 2
#define w 1
#define sizepop 4096
#define popmax 100
#define popmin -100
#define Vmax (popmax-popmin)
#define Vmin -(popmax-popmin)
#define gen 1000000
#define r (double)rand() / RAND_MAX  //-1 + 2 * ((double)rand()) / RAND_MAX
#define poprange popmax - popmin
// #define func(a) (((a)*(a)*(a)))
#define func(x) fabs(8000.0 + x * (-10000.0 + x * (-0.8 + x)));
hipDeviceProp_t prop;
double *pop=new double[sizepop];
double *V=new double[sizepop];
double *fitness=new double[sizepop];
double *pbest=new double[sizepop];
double *gbest=new double[1];
double *outputMax=new double[sizepop];
// double *answer=new double[1];
int best_index=0;
double *fitnesspbest=new double[sizepop];
double *fitnessgbest=new double[1];
// double elapsedTime;

// double func(double x) { return fabs(8000.0 + x * (-10000.0 + x * (-0.8 + x))); }

// extern double GPU_kernel(double *pop1, double *V1,double *fitnesspbest1,double *pbest1,double *gbest1,double *fitnessgbest1,double r1);


/*初始化粒子的位置和速度*/
void pop_init(double *pop){
        for(int i = 0; i < sizepop; i++){  

                pop[i] = r * poprange + popmin;  //position
                V[i] =  r * Vmax; 
                fitness[i] = func(pop[i]);   //value

        }
}

/*找出最大目標值*/
void max(double *fit,int size){
        int index = 0;
        double max = *fit; //set max value is first
     
        for(int i = 1;i < size; i++){
                if(*(fit+i) > max){
                    max = *(fit + i);
                    index = i;
                }
        }

        printf("max : %f\n",max);
        best_index = index;
        *fitnessgbest = max;
        outputMax[0] = max;
        
}


/*首先初始化粒子的位置和速度並求得「粒子自身最佳位置」、「群體最佳位置」、「個體最佳解」、「群體最佳解」*/
void PSO_init(void)
{
        pop_init(pop);
        max(fitness,sizepop);
        
        *gbest = pop[best_index];
        
        for(int i = 0; i < sizepop; i++){
                
                pbest[i] = pop[i];
                
        }
        for(int i = 0; i < sizepop; i++){
                fitnesspbest[i] = fitness[i];
        }

}

__global__ void kernel1(double *V, double *pop, double *pbest, double *gbest, double random1, double random2, double *fitness, double *fitnesspbest, double *outputMax){   //改用cuRand
        
        extern __shared__ double sdata[];
        int pno = blockIdx.x * blockDim.x +  threadIdx.x;
        int tid = threadIdx.x;

        if(fitnesspbest[pno] == outputMax[0]) *gbest = pbest[pno]; 
        __syncthreads(); 
        
        // if(tid == 0) printf("thread %d gbest: %lf\n",tid, *gbest);

        // printf(" %lf   %lf\n",random1,random2);
        // printf("V: %lf\n",V[i]);
        // printf("pno: %d \n",pno);
        /*update valocity*/
        V[pno] = w * V[pno] + c1 * (random1) * (pbest[pno] - pop[pno]) + c2 * (random2) * (*gbest - pop[pno]);   

        if (V[pno] > Vmax) V[pno] = Vmax;
        if (V[pno] < Vmin) V[pno] = Vmin;        

        // printf("pop: %lf\n",pop[i]);

        /*update position*/
        pop[pno] = pop[pno] + V[pno];
        
        if(pop[pno] > popmax) pop[pno] = popmax;
        if(pop[pno] < popmin) pop[pno] = popmin;

        /*update fitness*/
        fitness[pno] = func(pop[pno]); 

        /*update pbest、fitnesspbest*/
        if(fitness[pno] > fitnesspbest[pno]){       
                pbest[pno] = pop[pno];   
                fitnesspbest[pno] = fitness[pno]; 
        } 

        /*move fitnesspbest to shared memory data*/
        sdata[tid] = fitnesspbest[pno];
        __syncthreads();

        /*first interleaved reduction*/
        for(int s = 1; s < blockDim.x; s *= 2){

                int index = 2 * s * tid;
                if(index < blockDim.x){
                        sdata[index] = max(sdata[index], sdata[index + s]);
                }

                __syncthreads();
        }
        
        if(tid == 0) outputMax[blockIdx.x] = sdata[tid]; 



        
        
}
__global__ void kernel2(double *outputMax){   //利用reduction

        extern __shared__ double srdata[];
        int tid = threadIdx.x;
        srdata[tid] = outputMax[tid];
        __syncthreads();

        /*second reduction*/
        for(int s = 1; s < blockDim.x; s *= 2){

                int index = 2 * s * tid;
                if(index < blockDim.x){
                        srdata[index] = max(srdata[index], srdata[index + s]);
                }
                __syncthreads();
        }

        if(tid == 0) outputMax[tid] = srdata[tid];
        // if(tid == 0) printf("fitness gbest per iter: %lf\n", outputMax[0]); 

}

int main()
{       
        /*count time*/
        struct timespec start, end, temp;
        double time_used;
        clock_gettime(CLOCK_MONOTONIC, &start);

        hipGetDeviceProperties(&prop, DEV_NO);
        printf("maxThreadsPerBlock = %d\nsharedMemPerBlock = %lu\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);   //Shared memory available per block in bytes
        
        PSO_init(); 

	double *d_pop,*d_V,*d_fitnesspbest,*d_pbest;
        double *d_gbest,*d_fitnessgbest,*d_fitness;
        double *d_outputMax;
        
	// Allocate Memory Space on Device
        hipMalloc(&d_fitness,sizeof(double)*sizepop);
        hipMalloc(&d_pop,sizeof(double)*sizepop);
        hipMalloc(&d_V,sizeof(double)*sizepop);
        hipMalloc(&d_fitnesspbest,sizeof(double)*sizepop);
        hipMalloc(&d_pbest,sizeof(double)*sizepop);
        hipMalloc(&d_gbest,sizeof(double));
        hipMalloc(&d_fitnessgbest,sizeof(double));
        
        hipMalloc(&d_outputMax,sizeof(double) * ceil(((float)sizepop) / 1024));
 
	// Copy Data to be Calculated
        hipMemcpy(d_fitness, fitness, sizeof(double)*sizepop, hipMemcpyHostToDevice);
        hipMemcpy(d_pop, pop, sizeof(double)*sizepop, hipMemcpyHostToDevice);
        hipMemcpy(d_V, V, sizeof(double)*sizepop, hipMemcpyHostToDevice);
        hipMemcpy(d_fitnesspbest, fitnesspbest, sizeof(double)*sizepop, hipMemcpyHostToDevice);
        hipMemcpy(d_pbest, pbest, sizeof(double)*sizepop, hipMemcpyHostToDevice);
        hipMemcpy(d_gbest, gbest, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_fitnessgbest, fitnessgbest, sizeof(double), hipMemcpyHostToDevice);

        hipMemcpy(d_outputMax, outputMax, sizeof(double), hipMemcpyHostToDevice); 


        int num_block = ceil(((float)sizepop) / 1024);
        printf("number of block: %d\n",num_block);

        int num_threads = (sizepop < 1024)? sizepop : 1024;
        printf("number of threads: %d\n",num_threads);

        for(int i = 0; i < gen; i++){

                kernel1<<<num_block ,num_threads, num_threads * sizeof(double)>>>(d_V, d_pop, d_pbest, d_gbest, r, r, d_fitness, d_fitnesspbest, d_outputMax);
                kernel2<<<1, num_block, num_block * sizeof(double)>>>(d_outputMax);

        }

        /*final answer*/
        double answer;
        hipMemcpy(&answer, d_outputMax, sizeof(double), hipMemcpyDeviceToHost); 
        printf("The answer is: %lf\n", answer);

        /*等待kernel執行完*/
        // cudaDeviceSynchronize();

        /*count time*/
        clock_gettime(CLOCK_MONOTONIC, &end);
        if ((end.tv_nsec - start.tv_nsec) < 0) {
        temp.tv_sec = end.tv_sec-start.tv_sec-1;
        temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
        } else {
        temp.tv_sec = end.tv_sec - start.tv_sec;
        temp.tv_nsec = end.tv_nsec - start.tv_nsec;
        }
        time_used = temp.tv_sec + (double) temp.tv_nsec / 1000000000.0;
        
        printf("%f second\n", time_used);

}
