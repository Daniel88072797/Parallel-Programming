#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define Blockfactor 32
//======================
#define DEV_NO 0
hipDeviceProp_t prop;
const int INF = ((1 << 30) - 1);
const int V = 50010;

void input(char* inFileName);
void output(char* outFileName);

// void block_FW(int B);
int ceil(int a, int b);
// void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);

int n, m;
// static int Dist[V][V];

extern __shared__ int d2[];
__global__ void p1_cal(int B, int Round, int block_start_x, int block_start_y, int n ,int* Dist) {
    
    __shared__ int transpose[Blockfactor * (Blockfactor + 1)];

    int block_internal_start_x = block_start_x * B;
    int block_internal_start_y = block_start_y * B;


    int i = threadIdx.y + block_internal_start_x;
    int j = threadIdx.x + block_internal_start_y;
    if(i >= n) return;
    if(j >= n) return;
    d2[threadIdx.y * B + threadIdx.x] = Dist[i * n + j];
    // __syncthreads();

    for (int k = Round * B; k < (Round + 1) * B && k < n; ++k) {     
        __syncthreads();
        if (d2[threadIdx.y * B + (k - Round * B)] + d2[(k - Round * B) * B + threadIdx.x] < d2[threadIdx.y * B + threadIdx.x]) {
            d2[threadIdx.y * B + threadIdx.x] = d2[threadIdx.y * B + (k - Round * B)] + d2[(k - Round * B) * B + threadIdx.x];
        }

    }
    transpose[threadIdx.x * (B + 1) + threadIdx.y] = d2[threadIdx.y * B + threadIdx.x];
    
    Dist[i * n + j] = transpose[threadIdx.x * (B + 1) + threadIdx.y];
}


__global__ void p2_cal(int B, int Round, int block_start_x, int block_start_y, int n ,int* Dist, int c) {
    
    __shared__ int d1[Blockfactor];
    __shared__ int transpose[Blockfactor * (Blockfactor + 1)];

    int block_internal_start_x = (block_start_x + blockIdx.y) * B;
    int block_internal_start_y = (block_start_y + blockIdx.x) * B;

    //shared memory
    int i = threadIdx.y + block_internal_start_x;
    int j = threadIdx.x + block_internal_start_y;
    if(i >= n) return;
    if(j >= n) return;

    d2[threadIdx.y * B + threadIdx.x] = Dist[i * n + j];
    
    for (int k = Round * B; k < (Round + 1) * B && k < n; ++k) {  

        if(threadIdx.x == 0 && c == 1){
            d1[threadIdx.y] = Dist[i * n + k];
        }
        if(threadIdx.y == 0 && c == 2){
            d1[threadIdx.x] = Dist[k * n + j];
        }
        
        __syncthreads();
        // if (Dist[i * n + k] + Dist[k * n + j] < Dist[i * n + j]) {
        //     Dist[i * n + j] = Dist[i * n + k] + Dist[k * n + j];
        // }
        //d2[a * B + threadIdx.y]
        if(c == 1){
            if (d1[threadIdx.y] + d2[(k % B) * B + threadIdx.x] < d2[threadIdx.y * B + threadIdx.x]) {
                d2[threadIdx.y * B + threadIdx.x] = d1[threadIdx.y] + d2[(k % B) * B + threadIdx.x];
            }
        }
        else{
            if (d2[threadIdx.y * B + (k % B)] + d1[threadIdx.x] < d2[threadIdx.y * B + threadIdx.x]) {
                d2[threadIdx.y * B + threadIdx.x] = d2[threadIdx.y * B + (k % B)] + d1[threadIdx.x];
            }
        }
        __syncthreads();
     
    }

    transpose[threadIdx.x * (B + 1) + threadIdx.y] = d2[threadIdx.y * B + threadIdx.x];
    
    Dist[i * n + j] = transpose[threadIdx.x * (B + 1) + threadIdx.y];

}

__global__ void p3_cal(int B, int Round, int block_start_x, int block_start_y, int n ,int* Dist) {
    __shared__ int d0[Blockfactor];
    __shared__ int d1[Blockfactor];
    __shared__ int transpose[Blockfactor * (Blockfactor + 1)];

    int block_internal_start_x = (block_start_x + blockIdx.y) * B;
    int block_internal_start_y = (block_start_y + blockIdx.x) * B;
    //shared memory

    int i = threadIdx.y + block_internal_start_x;
    int j = threadIdx.x + block_internal_start_y;
    if(i >= n) return;
    if(j >= n) return;

    d2[threadIdx.y * B + threadIdx.x] = Dist[i * n + j];
    // __syncthreads();  

    for (int k = Round * B; k < (Round + 1) * B && k < n; ++k) {     

        // if(threadIdx.y == 0){
            d0[threadIdx.y] = Dist[i * n + k];
        // }
        // if(threadIdx.x == 0){
            d1[threadIdx.x] = Dist[k * n + j];
        // }
        
        // __syncthreads();

        if (d0[threadIdx.y] + d1[threadIdx.x] < d2[threadIdx.y * B + threadIdx.x]) {  //d0[threadIdx.x] + d1[threadIdx.y]
            d2[threadIdx.y * B + threadIdx.x] = d0[threadIdx.y] + d1[threadIdx.x];
        }
        

        // __syncthreads();
     
    }

    transpose[threadIdx.x * (B + 1) + threadIdx.y] = d2[threadIdx.y * B + threadIdx.x];
    
    Dist[i * n + j] = transpose[threadIdx.x * (B + 1) + threadIdx.y];
}


void block_FW(int B, int* Dist)
{


    int round = ceil(n, B);

    int *distance;
    hipMalloc(&distance, n * n * sizeof(int));
    // cudaMallocHost(&Dist, n * n * sizeof(int));
    hipMemcpy(distance, Dist, n * n * sizeof(int), hipMemcpyHostToDevice);

    // for(int i = 0; i < n * n; i++)
    // printf("Dist[%d] = %d\n", i, Dist[i]);

    dim3 num_threads(B, B);
    dim3 num_t(B, B + 1);
    
    
    // const int num_threads = B;
    const int sm_size = B * B * sizeof(int);
    for (int r = 0; r < round; ++r) {
        // printf("%d %d\n", r, round);
        //fflush(stdout);

        /* Phase 1*/
        
        p1_cal<<<1 ,num_threads, sm_size>>>(B, r, r, r, n, distance);

        dim3 num_block1(r, 1);
        dim3 num_block2(round - r - 1, 1);
        dim3 num_block3(1, r);
        dim3 num_block4(1, round - r - 1);
        /* Phase 2*/
        p2_cal<<<num_block1 ,num_threads, sm_size>>>(B, r, r, 0, n, distance, 1);
        p2_cal<<<num_block2 ,num_threads, sm_size>>>(B, r, r, r + 1, n, distance, 1);
        p2_cal<<<num_block3 ,num_threads, sm_size>>>(B, r, 0, r, n, distance, 2); //**
        p2_cal<<<num_block4 ,num_threads, sm_size>>>(B, r, r + 1, r, n, distance, 2); //** 

        dim3 num_block5(r, r);
        dim3 num_block6(round - r - 1, r);
        dim3 num_block7(r, round - r - 1);
        dim3 num_block8(round - r - 1, round - r - 1);
        /* Phase 3*/
        p3_cal<<<num_block5 ,num_threads, sm_size>>>(B, r, 0, 0, n, distance);
        p3_cal<<<num_block6 ,num_threads, sm_size>>>(B, r, 0, r + 1, n, distance);
        p3_cal<<<num_block7 ,num_threads, sm_size>>>(B, r, r + 1, 0, n, distance);
        p3_cal<<<num_block8 ,num_threads, sm_size>>>(B, r, r + 1, r + 1, n, distance);
    }

    hipMemcpy(Dist, distance, n * n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(distance);

}


int main(int argc, char* argv[]) {

    /*read the file*/
    
            
    FILE* file = fopen(argv[1], "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    int* Dist = (int *)malloc(n * n * sizeof(int));
    // printf("%d", n);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist[i * n + j] = 0;
            } else {
                Dist[i * n + j] = INF;
            }
        }
    }
    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * n + pair[1]] = pair[2];
    }
    fclose(file);
    // input(argv[1]);

     hipGetDeviceProperties(&prop, DEV_NO);
     printf("maxThreadsPerBlock = %d\nsharedMemPerBlock = %lu\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);   //Shared memory available per block in bytes
    
    int B = Blockfactor;
    block_FW(B, Dist);
    
    // output(argv[2]);

    /*save the file*/
    FILE* outfile = fopen(argv[2], "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (Dist[i * n + j] >= INF) Dist[i * n + j] = INF;
        }
        fwrite(Dist + i * n, sizeof(int), n, outfile);
    }
    fclose(outfile);
   
}

int ceil(int a, int b) { return (a + b - 1) / b; }





