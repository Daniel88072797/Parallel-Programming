#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#define Blockfactor 32
//======================
#define DEV_NO 0
hipDeviceProp_t prop;
const int INF = ((1 << 30) - 1);
const int V = 50010;

void input(char* inFileName);
void output(char* outFileName);

// void block_FW(int B);
int ceil(int a, int b);
// void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);

int n, m;
// static int Dist[V][V];

extern __shared__ int d2[];
__global__ void p1_cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int n ,int* Dist) {
    
    __shared__ int transpose[Blockfactor * (Blockfactor + 1)];
    int b_i = block_start_x;
    int b_j = block_start_y;
    int block_internal_start_x = b_i * B;

    int block_internal_start_y = b_j * B;


    int i = threadIdx.y + block_internal_start_x;
    int j = threadIdx.x + block_internal_start_y;
    if(i >= n) return;
    if(j >= n) return;
    d2[threadIdx.y * B + threadIdx.x] = Dist[i * n + j];
    // __syncthreads();
    int a = 0;

    
    for (int k = Round * B; k < (Round + 1) * B && k < n; ++k) {     
        __syncthreads();
        if (d2[threadIdx.y * B + a] + d2[a * B + threadIdx.x] < d2[threadIdx.y * B + threadIdx.x]) {
            d2[threadIdx.y * B + threadIdx.x] = d2[threadIdx.y * B + a] + d2[a * B + threadIdx.x];
        }

        a++;
        
    }
    transpose[threadIdx.x * (B + 1) + threadIdx.y] = d2[threadIdx.y * B + threadIdx.x];
    

    Dist[i * n + j] = transpose[threadIdx.x * (B + 1) + threadIdx.y];
}


__global__ void p2_cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int n ,int* Dist, int c) {
    
    __shared__ int d1[Blockfactor];
    __shared__ int transpose[Blockfactor * (Blockfactor + 1)];
    int b_i = block_start_x + blockIdx.y;
    int b_j = block_start_y + blockIdx.x;

    int block_internal_start_x = b_i * B;

    int block_internal_start_y = b_j * B;

    //shared memory

    int i = threadIdx.y + block_internal_start_x;
    int j = threadIdx.x + block_internal_start_y;
    if(i >= n) return;
    if(j >= n) return;

    d2[threadIdx.y * B + threadIdx.x] = Dist[i * n + j];
    
    
    for (int k = Round * B; k < (Round + 1) * B && k < n; ++k) {  

        if(threadIdx.x == 0 && c == 1){
            d1[threadIdx.y] = Dist[i * n + k];
        }
        if(threadIdx.y == 0 && c == 2){
            d1[threadIdx.x] = Dist[k * n + j];
        }
        
        __syncthreads();
        // if (Dist[i * n + k] + Dist[k * n + j] < Dist[i * n + j]) {
        //     Dist[i * n + j] = Dist[i * n + k] + Dist[k * n + j];
        // }
        //d2[a * B + threadIdx.y]
        if(c == 1){
            if (d1[threadIdx.y] + d2[(k % B) * B + threadIdx.x] < d2[threadIdx.y * B + threadIdx.x]) {
                d2[threadIdx.y * B + threadIdx.x] = d1[threadIdx.y] + d2[(k % B) * B + threadIdx.x];
            }
        }
        if(c == 2){
            if (d2[threadIdx.y * B + (k % B)] + d1[threadIdx.x] < d2[threadIdx.y * B + threadIdx.x]) {
                d2[threadIdx.y * B + threadIdx.x] = d2[threadIdx.y * B + (k % B)] + d1[threadIdx.x];
            }
        }
        __syncthreads();
     
    }

    transpose[threadIdx.x * (B + 1) + threadIdx.y] = d2[threadIdx.y * B + threadIdx.x];
    
    Dist[i * n + j] = transpose[threadIdx.x * (B + 1) + threadIdx.y];
    

}

__global__ void p3_cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int n ,int* Dist) {
    __shared__ int d0[Blockfactor];
    __shared__ int d1[Blockfactor];
    __shared__ int transpose[Blockfactor * (Blockfactor + 1)];
    int b_i = block_start_x + blockIdx.y;
    int b_j = block_start_y + blockIdx.x;
    int block_internal_start_x = b_i * B;
    int block_internal_start_y = b_j * B;
    //shared memory

    int i = threadIdx.y + block_internal_start_x;
    int j = threadIdx.x + block_internal_start_y;
    if(i >= n) return;
    if(j >= n) return;

    d2[threadIdx.y * B + threadIdx.x] = Dist[i * n + j];
    // __syncthreads();  




    for (int k = Round * B; k < (Round + 1) * B && k < n; ++k) {     

        // if(threadIdx.y == 0){
            d0[threadIdx.y] = Dist[i * n + k];
        // }
        // if(threadIdx.x == 0){
            d1[threadIdx.x] = Dist[k * n + j];
        // }
        
        // __syncthreads();

        if (d0[threadIdx.y] + d1[threadIdx.x] < d2[threadIdx.y * B + threadIdx.x]) {  //d0[threadIdx.x] + d1[threadIdx.y]
            d2[threadIdx.y * B + threadIdx.x] = d0[threadIdx.y] + d1[threadIdx.x];
        }
        

        // __syncthreads();
     
    }

    transpose[threadIdx.x * (B + 1) + threadIdx.y] = d2[threadIdx.y * B + threadIdx.x];
    
    Dist[i * n + j] = transpose[threadIdx.x * (B + 1) + threadIdx.y];
}



__global__ void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int n ,int* Dist) {

    int b_i = block_start_x + blockIdx.y;
    int b_j = block_start_y + blockIdx.x;
    int block_internal_start_x = b_i * B;
    int block_internal_start_y = b_j * B;
    //shared memory

    int i = threadIdx.x + block_internal_start_x;
    int j = threadIdx.y + block_internal_start_y;
    if(i >= n) return;
    if(j >= n) return;
        
    for (int k = Round * B; k < (Round + 1) * B && k < n; ++k) {     

        if (Dist[i * n + k] + Dist[k * n + j] < Dist[i * n + j]) {
            Dist[i * n + j] = Dist[i * n + k] + Dist[k * n + j];
        }
        

        __syncthreads();
     
    }


}

void block_FW(int B, int* Dist)
{


    int round = ceil(n, B);

    int* dev0Mem_Dist;
    int* dev1Mem_Dist;
    hipSetDevice(0);
    hipMalloc((void**)&dev0Mem_Dist, n * n * sizeof(int));

    hipSetDevice(1);
    hipMalloc((void**)&dev1Mem_Dist, n * n * sizeof(int));
    hipMemcpy(dev0Mem_Dist, Dist, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev1Mem_Dist, Dist, n * n * sizeof(int), hipMemcpyHostToDevice);



    // int *distance;
    // cudaMalloc(&distance, n * n * sizeof(int));
    // // cudaMallocHost(&Dist, n * n * sizeof(int));
    // cudaMemcpy(distance, Dist, n * n * sizeof(int), cudaMemcpyHostToDevice);

    // for(int i = 0; i < n * n; i++)
    // printf("Dist[%d] = %d\n", i, Dist[i]);

    dim3 num_threads(B, B);
    dim3 num_t(B, B + 1);
    
    
    // const int num_threads = B;
    const int sm_size = B * B * sizeof(int);
    for (int r = 0; r < round; ++r) {
        // printf("%d %d\n", r, round);
        //fflush(stdout);

        /* Phase 1*/
        hipSetDevice(0);
        p1_cal<<<1 ,num_threads, sm_size>>>(B, r, r, r, 1, 1, n, dev0Mem_Dist);
        hipMemcpyPeer(dev1Mem_Dist, 1, dev0Mem_Dist, 0, n * n * sizeof(int));
        dim3 num_block1(r, 1);
        dim3 num_block2(round - r - 1, 1);
        dim3 num_block3(1, r);
        dim3 num_block4(1, round - r - 1);

        #pragma omp parallel
        {
            
            unsigned int cpu_thread_id = omp_get_thread_num();
            unsigned int num_cpu_threads = omp_get_num_threads();
            hipSetDevice(cpu_thread_id);
            int gpu_id = -1;
            hipGetDevice(&gpu_id);
            printf("CPU thread %d (of %d) uses CUDA device %d\n",cpu_thread_id, num_cpu_threads, gpu_id);

            if(cpu_thread_id == 0){
                printf("%d", cpu_thread_id);
                hipSetDevice(0);
                p2_cal<<<num_block1 ,num_threads, sm_size>>>(B, r, r, 0, r, 1, n, dev0Mem_Dist, 1);
                p2_cal<<<num_block3 ,num_threads, sm_size>>>(B, r, 0, r, 1, r, n, dev0Mem_Dist, 2); //**
                hipMemcpyPeer(dev1Mem_Dist, 1, dev0Mem_Dist, 0, n * n * sizeof(int));
            }
            #pragma omp barrier
            if(cpu_thread_id == 1)
            {
                printf("%d", cpu_thread_id);
                hipSetDevice(1);
                
                p2_cal<<<num_block2 ,num_threads, sm_size>>>(B, r, r, r + 1, round - r - 1, 1, n, dev1Mem_Dist, 1);
                p2_cal<<<num_block4 ,num_threads, sm_size>>>(B, r, r + 1, r, 1, round - r - 1, n, dev1Mem_Dist, 2); //**
                hipMemcpyPeer(dev0Mem_Dist, 0, dev1Mem_Dist, 1,  n * n * sizeof(int));
            }
            
            



        }


        hipSetDevice(0);
        dim3 num_block5(r, r);
        dim3 num_block6(round - r - 1, r);
        dim3 num_block7(r, round - r - 1);
        dim3 num_block8(round - r - 1, round - r - 1);
        /* Phase 3*/
        p3_cal<<<num_block5 ,num_threads, sm_size>>>(B, r, 0, 0, r, r, n, dev0Mem_Dist);
        p3_cal<<<num_block6 ,num_threads, sm_size>>>(B, r, 0, r + 1, round - r - 1, r, n, dev0Mem_Dist);
        p3_cal<<<num_block7 ,num_threads, sm_size>>>(B, r, r + 1, 0, r, round - r - 1, n, dev0Mem_Dist);
        p3_cal<<<num_block8 ,num_threads, sm_size>>>(B, r, r + 1, r + 1, round - r - 1, round - r - 1, n, dev0Mem_Dist);
    }

    hipMemcpy(Dist, dev0Mem_Dist, n * n * sizeof(int), hipMemcpyDeviceToHost);
    // cudaFree(distance);

}


int main(int argc, char* argv[]) {

    /*read the file*/
    FILE* file = fopen(argv[1], "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);
    int* Dist = (int *)malloc(n * n * sizeof(int));
    printf("%d", n);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (i == j) {
                Dist[i * n + j] = 0;
            } else {
                Dist[i * n + j] = INF;
            }
        }
    }
    int pair[3];
    for (int i = 0; i < m; ++i) {
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * n + pair[1]] = pair[2];
    }
    fclose(file);
    // input(argv[1]);

    // cudaGetDeviceProperties(&prop, DEV_NO);
    // printf("maxThreadsPerBlock = %d\nsharedMemPerBlock = %lu\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);   //Shared memory available per block in bytes
    
    int B = Blockfactor;
    block_FW(B, Dist);
    
    // output(argv[2]);

    /*save the file*/
    FILE* outfile = fopen(argv[2], "w");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            if (Dist[i * n + j] >= INF) Dist[i * n + j] = INF;
        }
        fwrite(Dist + i * n, sizeof(int), n, outfile);
    }
    fclose(outfile);
   
}

int ceil(int a, int b) { return (a + b - 1) / b; }



// #pragma omp barrier
    
//     if(cpu_thread_id == 0){
//         printf("%d", cpu_thread_id);
//         cudaSetDevice(0);

//         cudaMemcpyPeer(dev1Mem_Dist, 1, dev0Mem_Dist, 0, n * n * sizeof(int));
//     }
//     if(cpu_thread_id == 1)
//     {
//         printf("%d", cpu_thread_id);
//         cudaSetDevice(1);

//         cudaMemcpyPeer(dev0Mem_Dist, 0, dev1Mem_Dist, 1,  n * n * sizeof(int));
//     }

// }

// #pragma omp parallel 
// {   
//     dim3 num_block1(r, 1);
//     dim3 num_block2(round - r - 1, 1);
//     dim3 num_block3(1, r);
//     dim3 num_block4(1, round - r - 1);
//     unsigned int cpu_thread_id = omp_get_thread_num();
//     if(cpu_thread_id == 0){
//         printf("%d", cpu_thread_id);
//         cudaSetDevice(0);
//         p2_cal<<<num_block1 ,num_threads, sm_size>>>(B, r, r, 0, r, 1, n, distance, 1);
//         p2_cal<<<num_block2 ,num_threads, sm_size>>>(B, r, r, r + 1, round - r - 1, 1, n, distance, 1);
//     }
//     if(cpu_thread_id == 1)
//     {
//         printf("%d", cpu_thread_id);
//         cudaSetDevice(0);
//         p2_cal<<<num_block3 ,num_threads, sm_size>>>(B, r, 0, r, 1, r, n, distance, 2); //**
//         p2_cal<<<num_block4 ,num_threads, sm_size>>>(B, r, r + 1, r, 1, round - r - 1, n, distance, 2); //**
//     }

// }